#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <array>
#include <chrono>
#include <glog/logging.h>

#define RETURN_IF_ERROR(error_code) if (error_code != hipSuccess) {LOG(ERROR) << "CUDA error:\r\ncode=" << error_code << ", name=" << hipGetErrorName(error_code);return error_code;}

#define RETURN_IF_NULL(p) if (p == nullptr) {LOG(ERROR) << "Host memory failed!"; exit( EXIT_FAILURE );}


__global__ void hello_from_gpu() {
    printf("Hello World from the the GPU\n");
    printf("How are you ?\n");
    const auto bid = blockIdx.x;
    const auto tid = threadIdx.x;
    const auto idx = tid + bid * blockDim.x;
    printf("bid:%d, tid:%d, idx:%d\n", bid, tid, idx);
}

hipError_t ErrorCheck(hipError_t error_code, const std::string &filename, int lineNumber) {
    if (error_code != hipSuccess) {
        printf("CUDA error:\r\ncode=%d, name=%s, description=%s\r\nfile=%s, line%d\r\n",
               error_code, hipGetErrorName(error_code), hipGetErrorString(error_code), filename, lineNumber);
        return error_code;
    }
    return error_code;
}

void HelloFromGPU() {
    hello_from_gpu<<<1, 4>>>();
    ErrorCheck(hipDeviceSynchronize(), __FILE__, __LINE__);
    LOG(INFO) << "Hello, World!";
}


hipError_t TestCudaDeviceCount() {
    int count;
    RETURN_IF_ERROR(hipGetDeviceCount(&count));
    LOG(INFO) << "Cuda device count: " << count;
    return hipSuccess;
}

hipError_t TestCudaDeviceProp() {
    hipDeviceProp_t prop;
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    int count;
    RETURN_IF_ERROR(hipGetDeviceCount(&count));
    for (int i = 0; i < count; ++i) {
        RETURN_IF_ERROR(hipGetDeviceProperties(&prop, 0));
        LOG(INFO) << "Cuda device name: " << prop.name;
    }
    int dev = 0;
    hipDeviceProp_t devProp;
    RETURN_IF_ERROR(hipGetDeviceProperties(&devProp, dev));
    std::cout << "使用GPU device " << dev << ": " << devProp.name << std::endl;
    std::cout << "SM的数量：" << devProp.multiProcessorCount << std::endl;
    std::cout << "每个线程块的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
    std::cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock << std::endl;
    std::cout << "每个EM的最大线程数：" << devProp.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "每个SM的最大线程束数：" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;

    return hipSuccess;
}

constexpr int N = 33 * 1024;

class TimeCost {
private:
    std::chrono::time_point <std::chrono::system_clock> start;
public:
    TimeCost() : start(std::chrono::system_clock::now()) {}

    ~TimeCost() = default;

    auto Cost() {
        // us
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - start);
        return duration.count();
    }

    auto reset() {
        start = std::chrono::system_clock::now();
    }
};

// used for device
template<typename T>
__global__ void vector_add(T *out, T *a, T *b, int n) {
    static_assert(std::is_arithmetic<T>::value, "T must be arithmetic type!");
    for (int i = 0; i < n; ++i) {
        out[i] = a[i] + b[i];
    }
}

template<typename T>
__global__ void vector_add2(T *out, T *a, T *b, int n) {
    static_assert(std::is_arithmetic<T>::value, "T must be arithmetic type!");
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n) {
        out[tid] = a[tid] + b[tid];
        tid += blockDim.x + gridDim.x;
    }
}

__global__ void vector_add3(float *a, float *b, float *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        c[i] = a[i] + b[i];
    }
}

hipError_t TestCudaVectorAdd() {
    // 1. allocate host and device memory
    int a[N], b[N], c[N];
    for (int i = 0; i < N; ++i) {
        a[i] = i * i;
        b[i] = -i;
    }

    int *d_a, *d_b, *d_c;
    RETURN_IF_ERROR(hipMalloc((void **) &d_a, sizeof(int) * N));
    RETURN_IF_ERROR(hipMalloc((void **) &d_b, sizeof(int) * N));
    RETURN_IF_ERROR(hipMalloc((void **) &d_c, sizeof(int) * N));

    // 2. copy data from host to device
    RETURN_IF_ERROR(hipMemcpy(d_a, a, sizeof(int) * N, hipMemcpyHostToDevice));
    RETURN_IF_ERROR(hipMemcpy(d_b, b, sizeof(int) * N, hipMemcpyHostToDevice));
    RETURN_IF_ERROR(hipMemcpy(d_c, c, sizeof(int) * N, hipMemcpyHostToDevice));

    // 3. launch kernel
    const int thread_blocks = 2, thread_per_block = 4;
    TimeCost cost{};
    for (int i = 0; i < N; ++i) {
        c[i] = a[i] + b[i];
    }
    std::cout << "Cost baseline: " << cost.Cost() << " us" << std::endl;
    cost.reset();

    vector_add<<<thread_blocks, thread_per_block>>>(d_c, d_a, d_b, N);
    std::cout << "Cost func1: " << cost.Cost() << " us" << std::endl;
    cost.reset();

    vector_add2<<<thread_blocks, thread_per_block>>>(d_c, d_a, d_b, N);
    std::cout << "Cost func2: " << cost.Cost() << " us" << std::endl;

    cost.reset();
    vector_add2<<<thread_blocks * 2, thread_per_block * 2>>>(d_c, d_a, d_b, N);
    std::cout << "Cost func3: " << cost.Cost() << " us" << std::endl;

    // 4. copy result from device to host
    RETURN_IF_ERROR(hipMemcpy(c, d_c, sizeof(int) * N, hipMemcpyDeviceToHost));
    std::cout << "\n";

    // 5. free device and host memory
    RETURN_IF_ERROR(hipFree(d_a));
    RETURN_IF_ERROR(hipFree(d_b));
    RETURN_IF_ERROR(hipFree(d_c));
    return hipSuccess;
}

constexpr int threadsPerBlock = 256;

__global__ void dot(float *a, float *b, float *c) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float temp = 0;
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    cache[cacheIndex] = temp;

    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + 1];
            __syncthreads();
        }
        i /= 2;
    }
    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}

#define DIM 1024

__global__ void blend_kernel(float *outSrc, const float *inSrc) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    int left = offset - 1;
    int right = offset + 1;
    if (x == 0) left++;
    if (x == DIM - 1) right++;

    int top = offset - DIM;
    int bottom = offset + DIM;
    if (y == 0) top += DIM;
    if (y == DIM - 1) bottom -= DIM;
}

__global__ void kernel1(uchar4 *ptr) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float fx = x / (float) DIM - 0.5f;
    float fy = y / (float) DIM - 0.5f;
    unsigned char green = 128 + 127 * sin(abs(fx * 100) - abs(fy * 100));
}

__global__ void histo_kernel(unsigned char *buffer, long size, unsigned int *histo) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    while (i < size) {
        atomicAdd(&(histo[buffer[i]]), 1);
        i += stride;
    }
}

__global__ void histo_kernel2(unsigned char *buffer, long size, unsigned int *histo) {
    __shared__ unsigned int temp[256];
    temp[threadIdx.x] = 0;
    __syncthreads();

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;
    while (i < size) {
        atomicAdd(&(temp[buffer[i]]), 1);
        i += offset;
    }

    __syncthreads();
    atomicAdd(&(histo[buffer[threadIdx.x]]), temp[threadIdx.x]);
}

int main() {

    // TestCudaDeviceCount();

    TestCudaDeviceProp();

    TestCudaVectorAdd();

    return 0;
}
