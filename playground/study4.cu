#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <array>
#include <chrono>
#include <type_traits>
#include <glog/logging.h>
#include <hip/hip_runtime.h>

#define RETURN_IF_ERROR(error_code) if (error_code != hipSuccess) {LOG(ERROR) << "CUDA error, code:" << error_code << ", name: " << hipGetErrorName(error_code);return error_code;}

#define RETURN_IF_NULL(p) if (p == nullptr) {LOG(ERROR) << "Host memory failed!"; exit( EXIT_FAILURE );}

class TimeCost {
private:
    std::chrono::time_point <std::chrono::system_clock> start;
public:
    TimeCost() : start(std::chrono::system_clock::now()) {}

    ~TimeCost() = default;

    auto Cost() {
        // us
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - start);
        return duration.count();
    }

    auto reset() {
        start = std::chrono::system_clock::now();
    }
};

inline hipError_t ErrorCheck(hipError_t error_code, const std::string &filename, int lineNumber) {
    if (error_code != hipSuccess) {
        printf("CUDA error:\r\ncode=%d, name=%s, description=%s\r\nfile=%s, line%d\r\n",
               error_code, hipGetErrorName(error_code), hipGetErrorString(error_code), filename, lineNumber);
        return error_code;
    }
    return error_code;
}

template<typename T, typename std::enable_if_t<std::is_arithmetic<T>::value, bool> = true>
__global__ void relu_kernel1(T *input, T *output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    return output[index] = input[index] < 0 ? 0 : input[index];
}

hipError_t TestCudaStream() {
    hipStream_t stream;
    RETURN_IF_ERROR(hipStreamCreate(&stream));

    constexpr int N = 1 << 8;
    float *d_a, *a;

    a = static_cast<float *>(malloc(N));
    RETURN_IF_NULL(a);

    RETURN_IF_ERROR(hipMalloc((void **) &d_a, N));
    TimeCost cost{};
    RETURN_IF_ERROR(hipMemcpyAsync((void *) d_a, (void *) a, N, hipMemcpyHostToDevice, stream));
    LOG(INFO) << "Async Copy Cost: " << cost.Cost() << " us";
    RETURN_IF_ERROR(hipStreamDestroy(stream));
    return hipSuccess;
}

__global__ void kernel_func1(float *a, int offset) {
    int index = blockDim.x * blockIdx.x + threadIdx.x + offset;
    auto x = static_cast<float>(a[index]);
    auto s = sinf(x), c = cosf(x);
    a[index] += sqrtf(s * s + c * s);
}

float maxError(float *a, int n) {
    float maxE = 0;
    for (int i = 0; i < n; i++) {
        float error = fabs(a[i] - 1.0f);
        if (error > maxE) maxE = error;
    }
    return maxE;
}

hipError_t TestAsyncStream() {
    constexpr int blockSize = 256, nStreams = 4;
    constexpr int n = 4 * 1024 * blockSize * nStreams;
    constexpr int streamSize = n / nStreams;
    constexpr int streamBytes = streamSize * sizeof(float);
    constexpr int bytes = n * sizeof(float);
    int devId = 0;

    hipDeviceProp_t prop;
    RETURN_IF_ERROR(hipGetDeviceProperties(&prop, devId));
    printf("Device : %s\n", prop.name);
    RETURN_IF_ERROR(hipSetDevice(devId));

    float *a, *d_a;
    RETURN_IF_ERROR(hipHostMalloc((void **) &a, bytes));
    RETURN_IF_ERROR(hipMalloc((void **) &d_a, bytes));

    // 1. sequence
    TimeCost cost{};
    memset((void *) a, 0, bytes);
    RETURN_IF_ERROR(hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice));
    kernel_func1<<<n / blockSize, blockSize>>>(d_a, 0);
    RETURN_IF_ERROR(hipMemcpy(a, d_a, bytes, hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    LOG(INFO) << "Seq Cost: " << cost.Cost() << " us";

    hipStream_t stream[nStreams];
    for (int i = 0; i < nStreams; ++i) {
        RETURN_IF_ERROR(hipStreamCreate(&stream[i]));
    }

    // 2. Async 1
    cost.reset();
    memset((void *) a, 0, bytes);
    for (int i = 0; i < nStreams; ++i) {
        int offset = streamSize * i;
        RETURN_IF_ERROR(hipMemcpyAsync(&d_a[offset], &a[offset], streamSize, hipMemcpyHostToDevice, stream[i]));
        kernel_func1<<<streamSize / blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
        RETURN_IF_ERROR(hipMemcpyAsync(&a[offset], &d_a[offset], streamSize, hipMemcpyDeviceToHost, stream[i]));
    }
    hipDeviceSynchronize();
    LOG(INFO) << "Async1 Cost: " << cost.Cost() << " us";

    // 3. Async 2
    cost.reset();
    memset((void *) a, 0, bytes);
    for (int i = 0; i < nStreams; ++i) {
        int offset = streamSize * i;
        RETURN_IF_ERROR(hipMemcpyAsync(&d_a[offset], &a[offset], streamSize, hipMemcpyHostToDevice, stream[i]));
    }
    for (int i = 0; i < nStreams; ++i) {
        int offset = streamSize * i;
        kernel_func1<<<streamSize / blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
    }
    for (int i = 0; i < nStreams; ++i) {
        int offset = streamSize * i;
        RETURN_IF_ERROR(hipMemcpyAsync(&a[offset], &d_a[offset], streamSize, hipMemcpyDeviceToHost, stream[i]));
    }
    hipDeviceSynchronize();
    LOG(INFO) << "Async2 Cost: " << cost.Cost() << " us";

    RETURN_IF_ERROR(hipHostFree(a));
    RETURN_IF_ERROR(hipFree(d_a));
    for (int i = 0; i < nStreams; ++i) {
        RETURN_IF_ERROR(hipStreamDestroy(stream[i]));
    }
    return hipSuccess;
}

// shared memory
__global__ void staticReverse(int *d, int n) {
    __shared__ int s[64];
    int t = threadIdx.x;
    int tr = n - t - 1;
    s[t] = d[t];
    __syncthreads();
    d[t] = s[tr];
}


__global__ void dynamicReverse(int *d, int n) {
    extern __shared__ int s[];
    int t = threadIdx.x;
    int tr = n - t - 1;
    s[t] = d[t];
    __syncthreads();
    d[t] = s[tr];
}

hipError_t TestReverse() {
    const int n = 64;
    int a[n], r[n], d[n];

    for (int i = 0; i < n; i++) {
        a[i] = i;
        r[i] = n - i - 1;
        d[i] = 0;
    }
    int *d_d;
    RETURN_IF_ERROR(hipMalloc((void **) &d_d, n * sizeof(int)));
    RETURN_IF_ERROR(hipMemcpy(d_d, d, n * sizeof(int), hipMemcpyHostToDevice));
    staticReverse<<<1, n>>>(d_d, n);
    RETURN_IF_ERROR(hipMemcpy(d, d_d, n * sizeof(int), hipMemcpyDeviceToHost));

    RETURN_IF_ERROR(hipMemcpy(d_d, d, n * sizeof(int), hipMemcpyHostToDevice));
    dynamicReverse<<<1, n, n * sizeof(int)>>>(d_d, n);
    RETURN_IF_ERROR(hipMemcpy(d, d_d, n * sizeof(int), hipMemcpyDeviceToHost));
    RETURN_IF_ERROR(hipFree(d_d));

    return hipSuccess;
}

// pinned memory
hipError_t TestPinnedMemory(){
    int * pinned_data, * portable_data;
    constexpr int N = 1024;
    // use pinned memory
    RETURN_IF_ERROR(hipHostMalloc((void**)&pinned_data,N*sizeof(int)));
    // don't use pinned memory
    portable_data = static_cast<int*>(malloc(N * sizeof(int)));
    return hipSuccess;
}

// Spaxy
__global__ void spaxy(int n, int a, float *x, float *y){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) y[index] = a * x[index] + x[index];
}

hipError_t TestSpaxy(){
    constexpr int N = 1 << 20;
    constexpr int block_size = 256;
    constexpr int grid_size = static_cast<int>((N + block_size -  1) / block_size);
    constexpr int a = 6;
    float *x, *y, *d_x, *d_y;
    hipHostMalloc(&x, N * sizeof (float ));
    hipHostMalloc(&y, N * sizeof (float ));
    hipMalloc(&d_x, N * sizeof(float ));
    hipMalloc(&d_y, N * sizeof(float ));
    memset(x, 0, N);
    memset(y, 0, N);
    hipMemcpy(d_x, x, N, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N, hipMemcpyHostToDevice);
    spaxy<<<grid_size, block_size>>>(N, 6, d_x, d_y);
    hipMemcpy(x, d_x, N, hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, N, hipMemcpyDeviceToHost);
    hipFree(d_x);
    hipFree(d_y);
    hipHostFree(x);
    hipHostFree(y);
    return hipSuccess;
}



int main() {
    TestCudaStream();

    TestAsyncStream();

    TestReverse();

    TestPinnedMemory();

    TestSpaxy();
}
